#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <stdio.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <n_events_cell.h>


template <typename scalar_t>
__global__ void group_by_bin_kernel(const scalar_t* events, 
                                    const int64_t* bins_count,
                                    int64_t batch_size, 
                                    int64_t event_size, 
                                    int64_t feature_size,
                                    int64_t n_intervals, 
                                    int64_t new_event_size,
                                    scalar_t* new_events) {

    const int batch_id = blockIdx.x * blockDim.x + threadIdx.x;
	const int interval_id = blockIdx.y * blockDim.y + threadIdx.y;
	const int event_id = blockIdx.z * blockDim.z + threadIdx.z;

	if (batch_id < batch_size & interval_id < n_intervals & event_id < new_event_size) {
	    // bins_count[batch_id][interval_id]
        int interval_len = bins_count[batch_id * n_intervals + interval_id];
	    if (event_id < interval_len){
            int offset = 0;
            for (int i = 0; i < interval_id; i++){
                // bins_count[batch_id][i]
                offset += bins_count[batch_id * n_intervals + i];
            }
            if ((event_id + offset) < event_size) {
                auto write_offset = batch_id * n_intervals * (new_event_size * feature_size) + \
                                    interval_id * (new_event_size * feature_size) + \
                                    event_id * feature_size;
                auto read_offset = batch_id * (event_size * feature_size) + offset * feature_size + \
                                   event_id * feature_size;

                for (int f = 0; f < feature_size; f++){
                    // new_events[batch_id + interval_id * n_intervals]
                    new_events[write_offset + f] = events[read_offset + f];
                }
            }
        }
	}
}


torch::Tensor group_by_bin_wrapper(torch::Tensor events,
                                   torch::Tensor bins_count){

    // events.shape = [batch_size, n_events, features]
	const auto batch_size = events.size(0);
	const auto event_size = events.size(1);
	const auto feature_size = events.size(2);
	// bins_count.shape = [batch_size, n_intervals]
	const auto n_intervals = bins_count.size(1);

    const auto new_event_size = torch::max(bins_count).cpu().item().to<int64_t>();
	auto new_events = torch::zeros({batch_size * n_intervals,
                                    new_event_size, feature_size}, events.options());

	dim3 threadsPerBlock(4, 4, 64);
    dim3 numBlocks((int)((batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x),
                   (int)((n_intervals + threadsPerBlock.y - 1) / threadsPerBlock.y),
                   (int)((new_event_size + threadsPerBlock.z - 1) / threadsPerBlock.z));

	AT_DISPATCH_ALL_TYPES(events.type(), "group_by_bin", ([&] {
		group_by_bin_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
			events.data<scalar_t>(),
			bins_count.data<int64_t>(),
			batch_size,
			event_size,
			feature_size,
			n_intervals,
			new_event_size,
			new_events.data<scalar_t>());
	}));

	return new_events;
}
